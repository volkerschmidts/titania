
#include <cuda_helper.hpp>
#include <iostream>
std::string get_gpu_device_name ( int count )
{
   hipDeviceProp_t deviceProp;
   hipGetDeviceProperties (&deviceProp, count);
   hipDeviceReset();
   return deviceProp.name;
}

double get_gpu_memory ( int count )
{
   hipDeviceProp_t deviceProp;
   hipGetDeviceProperties (&deviceProp, count);
   hipDeviceReset();
   return (deviceProp.totalGlobalMem / (1024.0*1024.0));
}

int get_number_of_gpu_devices()
{
   int count;
   hipError_t error_code = hipGetDeviceCount(&count);
   if ( error_code ) return 0;
   hipDeviceReset();
   return count;
}
