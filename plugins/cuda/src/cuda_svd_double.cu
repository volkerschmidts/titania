#include "hip/hip_runtime.h"

#include "../../cuda/include/cuda_svd_double.hpp"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include "../include/cuda_svd.hpp"
#include "../include/cuda_helper.hpp"


int cuda_double_svd( double *A, double *S, double *U, double *V, int m, int n, int lda, bool reset )
{
   hipError_t cudaStat1 = hipSuccess;
   hipError_t cudaStat2 = hipSuccess;
   hipError_t cudaStat3 = hipSuccess;
   hipError_t cudaStat4 = hipSuccess;
   hipError_t cudaStat5 = hipSuccess;

   // Device pointers
   double *d_A, *d_S, *d_U, *d_V;
   constexpr size_t size_d = sizeof(double);
   d_A = d_S = d_U = d_V = NULL;

   const long unsigned int S_size   = get_size_(  1, min(m,n), size_d);
   const long unsigned int U_size   = get_size_(  m, min(m,n), size_d);
   const long unsigned int V_size   = get_size_(  n, min(m,n), size_d);
   const long unsigned int A_size   = get_size_(lda,       n , size_d);

   // Allocate memory on GPU
   cudaStat1 = hipMalloc ((void**)&d_A   , A_size);
   cudaStat2 = hipMalloc ((void**)&d_S   , S_size);
   cudaStat3 = hipMalloc ((void**)&d_U   , U_size);
   cudaStat4 = hipMalloc ((void**)&d_V   , V_size);

   if (hipSuccess != cudaStat1) return ERROR_CUDA_MALLOC;
   if (hipSuccess != cudaStat2) return ERROR_CUDA_MALLOC;
   if (hipSuccess != cudaStat3) return ERROR_CUDA_MALLOC;
   if (hipSuccess != cudaStat4) return ERROR_CUDA_MALLOC;

   // Copy initial matrix on GPU
   cudaStat1 = hipMemcpy(d_A, A, A_size, hipMemcpyHostToDevice);
   
   if (hipSuccess != cudaStat1)  return ERROR_CUDA_COPY;

   cuda_double_svd_device_memory( d_A, d_S, d_U, d_V, m, n, lda );

   // Copy results
   cudaStat1 = hipMemcpy(U, d_U, U_size, hipMemcpyDeviceToHost);
   cudaStat2 = hipMemcpy(V, d_V, V_size, hipMemcpyDeviceToHost);
   cudaStat3 = hipMemcpy(S, d_S, S_size, hipMemcpyDeviceToHost);
   cudaStat5 = hipDeviceSynchronize();

   if (hipSuccess != cudaStat1) return ERROR_CUDA_COPY;
   if (hipSuccess != cudaStat2) return ERROR_CUDA_COPY;
   if (hipSuccess != cudaStat3) return ERROR_CUDA_COPY;
   if (hipSuccess != cudaStat4) return ERROR_CUDA_COPY;
   if (hipSuccess != cudaStat5) return ERROR_CUDA_COPY;

// Free device memory
   if (d_A    ) hipFree(d_A);
   if (d_S    ) hipFree(d_S);
   if (d_U    ) hipFree(d_U);
   if (d_V    ) hipFree(d_V);

   if ( reset ) hipDeviceReset();
   return 0;
}

int cuda_double_svd_device_memory( double *d_A, double *d_S, double *d_U, double *d_V, int m, int n, int lda )
{
   hipsolverHandle_t cusolverH = NULL;
   hipStream_t stream = NULL;

   hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
   hipError_t cudaStat = hipSuccess;

   // Device pointers
   double *d_work;
   d_work = NULL;

   int info, return_value;
   int *d_info = NULL;  /* error info */
   int lwork = 0;       /* size of workspace */

   return_value = 0;

   // Create cusolver handle
   status = hipsolverDnCreate(&cusolverH);
   if (HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_CUSOLVER_DN_CREATE;

   cudaStat = hipMalloc ((void**)&d_info, sizeof(int));
   if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }

   // Querry buffer
   status = hipsolverDnDgesvd_bufferSize( cusolverH, m, n, &lwork );
   if ( HIPSOLVER_STATUS_SUCCESS != status ) { return_value = ERROR_CUDA_BUFFER_SIZE; goto cleanup; }

   // Allocate workspace
   cudaStat = hipMalloc((void**)&d_work , sizeof(double)*lwork);
   if (hipSuccess != cudaStat ) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }

   // Compute SVD
   status = hipsolverDnDgesvd (
                              cusolverH,
                              'S',        // compute vectors in the range of A
                              'S',        // compute vectors in the range of A
                              m,
                              n,
                              d_A,
                              lda,
                              d_S,
                              d_U,
                              m,
                              d_V,
                              n,          // The lead dimension is smaller due to the fact that we calculate reduced vectors
                              d_work,
                              lwork,
                              NULL,
                              d_info
                             );

   // wait for the calculation
   cudaStat = hipDeviceSynchronize();
   if ( hipSuccess != cudaStat ) { std::cout << "cudastat = " << cudaStat << std::endl; return_value = ERROR_CUDA_DEVICE_SYNCHRONIZE; goto cleanup; }
   if ( HIPSOLVER_STATUS_SUCCESS != status )
   {
      cudaStat = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
      if (hipSuccess == cudaStat)
      {
         if ( info ) std::cerr << "ERROR:\tcudasolverDnDgesvd error code: " << info << std::endl;
      }
      else
      {
         std::cerr << "ERROR:\tcudasolverDnDgesvd failed. TITANIA was not able to get addional information on the\n"
                   << "\tproblem via cudasolverDnDgesvd info...\n";
      }
      return_value = ERROR_CUDA_SVD_S;
   }

   cleanup:
      if (d_work ) hipFree(d_work);
      if (cusolverH) hipsolverDnDestroy(cusolverH);
      if (stream   ) hipStreamDestroy(stream);
      return return_value;
}
