
#include <cuda_get_information.hpp>
#include <iostream>


void cuda_print_information()
{
   hipDeviceProp_t deviceProp;
   hipGetDeviceProperties (&deviceProp, 0);

   int i;

   std::cout << "ECCEnabled  : " << deviceProp.ECCEnabled << std::endl;
   std::cout << "asyncEngineCount  : " << deviceProp.asyncEngineCount << std::endl;
   std::cout << "canMapHostMemory  : " << deviceProp.canMapHostMemory << std::endl;
   std::cout << "canUseHostPointerForRegisteredMem  : " << deviceProp.canUseHostPointerForRegisteredMem << std::endl;
   std::cout << "clockRate  : " << deviceProp.clockRate << std::endl;
   std::cout << "computeMode  : " << deviceProp.computeMode << std::endl;
   std::cout << "computePreemptionSupported  : " << deviceProp.computePreemptionSupported << std::endl;
   std::cout << "concurrentKernels  : " << deviceProp.concurrentKernels << std::endl;
   std::cout << "concurrentManagedAccess  : " << deviceProp.concurrentManagedAccess << std::endl;
   std::cout << "cooperativeLaunch  : " << deviceProp.cooperativeLaunch << std::endl;
   std::cout << "cooperativeMultiDeviceLaunch  : " << deviceProp.cooperativeMultiDeviceLaunch << std::endl;
   std::cout << "deviceOverlap  : " << deviceProp.deviceOverlap << std::endl;
   std::cout << "directManagedMemAccessFromHost  : " << deviceProp.directManagedMemAccessFromHost << std::endl;
   std::cout << "globalL1CacheSupported  : " << deviceProp.globalL1CacheSupported << std::endl;
   std::cout << "hostNativeAtomicSupported  : " << deviceProp.hostNativeAtomicSupported << std::endl;
   std::cout << "integrated  : " << deviceProp.integrated << std::endl;
   std::cout << "isMultiGpuBoard  : " << deviceProp.isMultiGpuBoard << std::endl;
   std::cout << "kernelExecTimeoutEnabled  : " << deviceProp.kernelExecTimeoutEnabled << std::endl;
   std::cout << "l2CacheSize  : " << deviceProp.l2CacheSize / ( 1024.0)<< std::endl;
   std::cout << "localL1CacheSupported  : " << deviceProp.localL1CacheSupported << std::endl;
   std::cout << "luid  : " << deviceProp.luid << std::endl;
   std::cout << "luidDeviceNodeMask  : " << deviceProp.luidDeviceNodeMask << std::endl;
   std::cout << "major  : " << deviceProp.major << std::endl;
   std::cout << "managedMemory  : " << deviceProp.managedMemory << std::endl;

   for ( i = 0; i < 3; ++i )
   std::cout << "maxGridSize  : " << deviceProp.maxGridSize[i] << std::endl;

   std::cout << "maxSurface1D  : " << deviceProp.maxSurface1D << std::endl;

   for ( i = 0; i < 2; ++i )
   std::cout << "maxSurface1DLayered  : " << deviceProp.maxSurface1DLayered[i] << std::endl;

   for ( i = 0; i < 2; ++i )
   std::cout << "maxSurface2D  : " << deviceProp.maxSurface2D[i] << std::endl;

   for ( i = 0; i < 3; ++i )
   std::cout << "maxSurface2DLayered  : " << deviceProp.maxSurface2DLayered[i] << std::endl;

   for ( i = 0; i < 3; ++i )
   std::cout << "maxSurface3D  : " << deviceProp.maxSurface3D[i] << std::endl;

   std::cout << "maxSurfaceCubemap  : " << deviceProp.maxSurfaceCubemap << std::endl;

   for ( i = 0; i < i; ++i )
   std::cout << "maxSurfaceCubemapLayered  : " << deviceProp.maxSurfaceCubemapLayered[i] << std::endl;

   std::cout << "maxTexture1D  : " << deviceProp.maxTexture1D << std::endl;

   for ( i = 0; i < i; ++i )
   std::cout << "maxTexture1DLayered  : " << deviceProp.maxTexture1DLayered[i] << std::endl;

   std::cout << "maxTexture1DLinear  : " << deviceProp.maxTexture1DLinear << std::endl;
   std::cout << "maxTexture1DMipmap  : " << deviceProp.maxTexture1DMipmap << std::endl;

   for ( i = 0; i < 2; ++i )
   std::cout << "maxTexture2D  : " << deviceProp.maxTexture2D[i] << std::endl;

   for ( i = 0; i < 2; ++i )
   std::cout << "maxTexture2DGather  : " << deviceProp.maxTexture2DGather[i] << std::endl;

   for ( i = 0; i < 3; ++i )
   std::cout << "maxTexture2DLayered  : " << deviceProp.maxTexture2DLayered[i] << std::endl;

   for ( i = 0; i < 3; ++i )
   std::cout << "maxTexture2DLinear  : " << deviceProp.maxTexture2DLinear[i] << std::endl;

   for ( i = 0; i < 2; ++i )
   std::cout << "maxTexture2DMipmap  : " << deviceProp.maxTexture2DMipmap[i] << std::endl;

   for ( i = 0; i < 3; ++i )
   std::cout << "maxTexture3D  : " << deviceProp.maxTexture3D[i] << std::endl;

   for ( i = 0; i < 3; ++i )
   std::cout << "maxTexture3DAlt  : " << deviceProp.maxTexture3DAlt[i] << std::endl;

   std::cout << "maxTextureCubemap  : " << deviceProp.maxTextureCubemap << std::endl;

   for ( i = 0; i < 2; ++i )
   std::cout << "maxTextureCubemapLayered  : " << deviceProp.maxTextureCubemapLayered[i] << std::endl;

   for ( i = 0; i < 3; ++i )
   std::cout << "maxThreadsDim  : " << deviceProp.maxThreadsDim[i] << std::endl;

   std::cout << "maxThreadsPerBlock  : " << deviceProp.maxThreadsPerBlock << std::endl;
   std::cout << "maxThreadsPerMultiProcessor  : " << deviceProp.maxThreadsPerMultiProcessor << std::endl;

   std::cout << "memPitch  : " << deviceProp.memPitch << std::endl;
   std::cout << "memoryBusWidth  : " << deviceProp.memoryBusWidth << std::endl;
   std::cout << "memoryClockRate  : " << deviceProp.memoryClockRate << std::endl;
   std::cout << "minor  : " << deviceProp.minor << std::endl;
   std::cout << "multiGpuBoardGroupID  : " << deviceProp.multiGpuBoardGroupID << std::endl;
   std::cout << "multiProcessorCount  : " << deviceProp.multiProcessorCount << std::endl;
   std::cout << "name  : " << deviceProp.name << std::endl;
   std::cout << "pageableMemoryAccess  : " << deviceProp.pageableMemoryAccess << std::endl;
   std::cout << "pageableMemoryAccessUsesHostPageTables  : " << deviceProp.pageableMemoryAccessUsesHostPageTables << std::endl;
   std::cout << "pciBusID  : " << deviceProp.pciBusID << std::endl;
   std::cout << "pciDeviceID  : " << deviceProp.pciDeviceID << std::endl;
   std::cout << "pciDomainID  : " << deviceProp.pciDomainID << std::endl;
   std::cout << "regsPerBlock  : " << deviceProp.regsPerBlock << std::endl;
   std::cout << "regsPerMultiprocessor  : " << deviceProp.regsPerMultiprocessor << std::endl;
   std::cout << "sharedMemPerBlock  : " << deviceProp.sharedMemPerBlock << std::endl;
   std::cout << "sharedMemPerBlockOptin  : " << deviceProp.sharedMemPerBlockOptin << std::endl;
   std::cout << "sharedMemPerMultiprocessor  : " << deviceProp.sharedMemPerMultiprocessor << std::endl;
   std::cout << "singleToDoublePrecisionPerfRatio  : " << deviceProp.singleToDoublePrecisionPerfRatio << std::endl;
   std::cout << "streamPrioritiesSupported  : " << deviceProp.streamPrioritiesSupported << std::endl;
   std::cout << "surfaceAlignment  : " << deviceProp.surfaceAlignment << std::endl;
   std::cout << "tccDriver  : " << deviceProp.tccDriver << std::endl;
   std::cout << "textureAlignment  : " << deviceProp.textureAlignment << std::endl;
   std::cout << "texturePitchAlignment  : " << deviceProp.texturePitchAlignment << std::endl;
   std::cout << "totalConstMem  : " << deviceProp.totalConstMem << std::endl;
   std::cout << "totalGlobalMem  : " << deviceProp.totalGlobalMem / ( 1024.0*1024.0 )<< std::endl;
   std::cout << "unifiedAddressing  : " << deviceProp.unifiedAddressing << std::endl;
   //std::cout << "uuid  : " << deviceProp.uuid << std::endl;
   std::cout << "warpSize  : " << deviceProp.warpSize << std::endl;
}


void cuda_print_memory_information()
{
   constexpr double bytes_2_mb = ( 1024.0*1024.0 );
   hipDeviceProp_t deviceProp;
   hipGetDeviceProperties (&deviceProp, 0);

   size_t free, total;
   //std::cout << "Device name: " << deviceProp.name << std::endl;
   hipMemGetInfo(&free,&total);
   std::cout << "Free memory: " << free/ bytes_2_mb << " MB\nTotal memory: " << total/ bytes_2_mb << " MB" << std::endl;
}
