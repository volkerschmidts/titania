
#include "../../cuda/include/cuda_svd_single.hpp"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "../../cuda/include/cuda_svd.hpp"

/*constexpr float mb_fac = 1024.0*1024.0;

void check_memory()
{
   size_t free_byte ;
   size_t total_byte ;
   hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
   if ( hipSuccess != cuda_status )
   {
      printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
      exit(1);
   }

   float free_db = (float)free_byte ;
   float total_db = (float)total_byte ;
   float used_db = total_db - free_db ;

   printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n", 
           used_db/mb_fac, 
           free_db/mb_fac, 
           total_db/mb_fac);
}*/

int cuda_single_svd( float *A, float *S, float *U, float *V, int m, int n, int lda )
{
   hipsolverHandle_t cusolverH = NULL;
   hipStream_t stream = NULL;
   hipsolverGesvdjInfo_t gesvdj_params = NULL;

   hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
   hipError_t cudaStat1 = hipSuccess;
   hipError_t cudaStat2 = hipSuccess;
   hipError_t cudaStat3 = hipSuccess;
   hipError_t cudaStat4 = hipSuccess;
   hipError_t cudaStat5 = hipSuccess;

   // Device pointers
   float *d_work, *d_A, *d_S, *d_U, *d_V;
   d_work = d_A = d_S = d_U = d_V = NULL;

   int *d_info = NULL;  /* error info */
   int lwork = 0;       /* size of workspace */

   // Configuration of gesvdj
   const float tol = 1.e-7;
   const int econ = 0; /* econ = 1 for economy size */

   // Numerical results of gesvdj
//   double residual = 0;
//   int executed_sweeps = 0;

   // Create cusolver handle
   status = hipsolverDnCreate(&cusolverH);
   if (HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_CUSOLVER_DN_CREATE;

   cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
   if ( hipSuccess != cudaStat1 ) return ERROR_STREAM_CREATE_WITH_FLAGS;
   
   // Bind a stream
   status = hipsolverSetStream(cusolverH, stream);
   if ( HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_SET_STREAM;

   // Configure gesvdj 
   status = hipsolverDnCreateGesvdjInfo(&gesvdj_params);
   if ( HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_CREATE_SVD_INFO;

   // Set tolerance
   status = hipsolverDnXgesvdjSetTolerance( gesvdj_params, tol);
   if ( HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_SET_SVD_PARAMETER;

   // Set sweeps
//   status = hipsolverDnXgesvdjSetMaxSweeps( gesvdj_params, max_sweeps);
//   if ( HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_SET_SVD_PARAMETER;

   // Allocate memory on GPU
   cudaStat1 = hipMalloc ((void**)&d_A   , sizeof(float)*lda*n);
   cudaStat2 = hipMalloc ((void**)&d_S   , sizeof(float)*n);
   cudaStat3 = hipMalloc ((void**)&d_U   , sizeof(float)*lda*m);
   cudaStat4 = hipMalloc ((void**)&d_V   , sizeof(float)*lda*n);
   cudaStat5 = hipMalloc ((void**)&d_info, sizeof(int));
   
   if (hipSuccess != cudaStat1) return ERROR_CUDA_MALLOC;
   if (hipSuccess != cudaStat2) return ERROR_CUDA_MALLOC;
   if (hipSuccess != cudaStat3) return ERROR_CUDA_MALLOC;
   if (hipSuccess != cudaStat4) return ERROR_CUDA_MALLOC;
   if (hipSuccess != cudaStat5) return ERROR_CUDA_MALLOC;

   // Copy initial matrix on GPU
   cudaStat1 = hipMemcpy(d_A, A, sizeof(float)*lda*n, hipMemcpyHostToDevice);
   
   if (hipSuccess != cudaStat1)  return ERROR_CUDA_COPY;
 
   // Querry buffer
   status = hipsolverDnSgesvdj_bufferSize(
                                         cusolverH,
                                         HIPSOLVER_EIG_MODE_VECTOR,
                                         econ,
                                         m,
                                         n,
                                         d_A,
                                         lda,
                                         d_S,
                                         d_U,
                                         lda,
                                         d_V,
                                         lda,
                                         &lwork,
                                         gesvdj_params);
   if ( HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_CUDA_BUFFER_SIZE;

   // Allocate workspace
   cudaStat1 = hipMalloc((void**)&d_work , sizeof(float)*lwork);
   if (hipSuccess != cudaStat1 ) return ERROR_CUDA_MALLOC;

   // Compute SVD
   status = hipsolverDnSgesvdj(
                              cusolverH,
                              HIPSOLVER_EIG_MODE_VECTOR,
                              econ,
                              m,
                              n,
                              d_A,
                              lda,
                              d_S,
                              d_U,
                              lda,
                              d_V,
                              lda,
                              d_work,
                              lwork,
                              d_info,
                              gesvdj_params);
   // wait for the calculation
   cudaStat1 = hipDeviceSynchronize();
   if ( HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_CUDA_SVD_S;
   if ( hipSuccess != cudaStat1 ) return ERROR_CUDA_DEVICE_SYNCHRONIZE;

   // Copy results
   cudaStat1 = hipMemcpy(U, d_U, sizeof(float)*lda*m, hipMemcpyDeviceToHost);
   cudaStat2 = hipMemcpy(V, d_V, sizeof(float)*lda*n, hipMemcpyDeviceToHost);
   cudaStat3 = hipMemcpy(S, d_S, sizeof(float)*n    , hipMemcpyDeviceToHost);
   cudaStat5 = hipDeviceSynchronize();

   if (hipSuccess != cudaStat1) return ERROR_CUDA_COPY;
   if (hipSuccess != cudaStat2) return ERROR_CUDA_COPY;
   if (hipSuccess != cudaStat3) return ERROR_CUDA_COPY;
   if (hipSuccess != cudaStat4) return ERROR_CUDA_COPY;
   if (hipSuccess != cudaStat5) return ERROR_CUDA_COPY;

//   status = hipsolverDnXgesvdjGetSweeps( cusolverH, gesvdj_params, &executed_sweeps);
//   if ( HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_READ_INFORMATION;

//   status = hipsolverDnXgesvdjGetResidual( cusolverH, gesvdj_params, &residual);
//   if (HIPSOLVER_STATUS_SUCCESS != status) return ERROR_READ_INFORMATION;

// Free device memory
   if (d_A    ) hipFree(d_A);
   if (d_S    ) hipFree(d_S);
   if (d_U    ) hipFree(d_U);
   if (d_V    ) hipFree(d_V);
   if (d_work ) hipFree(d_work);

   if (cusolverH) hipsolverDnDestroy(cusolverH);
   if (stream      ) hipStreamDestroy(stream);
   if (gesvdj_params) hipsolverDnDestroyGesvdjInfo(gesvdj_params);

   hipDeviceReset();
   return 0;
}
