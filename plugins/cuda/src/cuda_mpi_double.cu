#include "hip/hip_runtime.h"

#include "../include/cuda_mpi_double.hpp"

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <iostream>

#include "../include/cuda_matmult_double.hpp"
#include "../include/cuda_svd.hpp"
#include "../include/cuda_svd_double.hpp"
#include "../include/cuda_get_information.hpp"
#include "../include/cuda_helper.hpp"

#define THREADS_PER_BLOCK 512

   constexpr size_t size_d = sizeof(double);

__global__ void cuda_invert_singular_values_device_memory( double *d_S, double *d_S_inv, const double cutoff, const int num_of_sigma, const int num_of_el )
{
   int index = threadIdx.x + blockIdx.x * blockDim.x;
   if ( index < num_of_el )
   {
      int sigma_index = ( index / num_of_sigma );
      if ( sigma_index >= num_of_sigma ) d_S_inv[index] = .0;
      else
      {
         bool is_sigma = (( index - sigma_index ) % num_of_sigma ) == 0;
         if ( is_sigma && d_S[sigma_index] > cutoff ) d_S_inv[index] = ( 1.0 / d_S[sigma_index] );
         else d_S_inv[index] = .0;
      }
   }
}

void printDeviceMatrix ( double* M, int m, int n, int lda )
{
   double *tmp = (double*) malloc( m*n*sizeof(double));
   hipMemcpy(tmp, M, lda*n*sizeof(double),hipMemcpyDeviceToHost);
   for ( int r = 0; r < m; ++r )
   {
      for ( int c = 0; c < n; ++c ) std::cout << tmp[r+c*lda] << "  ";
      std::cout << std::endl;
   }
   free(tmp);
}

int check_needed_memory ( int lda, int m, int n, size_t type_size )
{
   // Calculate the needed space on dRAM if all matrizes are kept on it,
   // but are deleted as soon as they are not needed anymore.
   long unsigned int requested_matrizes = type_size * n * ( m + 3*n );

   // Calculate the needed space on dRAM if S_inv and U share the memory
   // since they are the largest matrizes. U will be buffered on main RAM.
   long unsigned int reducable_size = type_size * n * ( m + n +1 );

   // Additional worksize of cusolverDNDgesvd
   hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
   hipsolverHandle_t cusolverH = NULL;

   int work = 0;
   long unsigned int free_i;
   size_t free_s, total;

   // Create cusolver handle
   status = hipsolverDnCreate(&cusolverH);
   if (HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_CUSOLVER_DN_CREATE;

   // Querry buffer
   status = hipsolverDnDgesvd_bufferSize( cusolverH, m, n, &work );
   if ( HIPSOLVER_STATUS_SUCCESS != status ) return ERROR_CUDA_BUFFER_SIZE;

   // Run cudaMemGetInfor here since cuslverDnCreate allocates
   // space for the cusolverH
   hipMemGetInfo(&free_s,&total);

   if (cusolverH) hipsolverDnDestroy(cusolverH);

   work *= type_size;
   requested_matrizes += ((long unsigned int) work);
   reducable_size += ((long unsigned int) work);

   free_i = (long unsigned int) free_s;

   if ( free_i > requested_matrizes && (free_i - requested_matrizes) > CUDA_SAVE_MEMORY_BUFFER_ ) return 0;
   else if ( free_i > reducable_size && (free_i - reducable_size) > CUDA_SAVE_MEMORY_BUFFER_ ) return 1;

   return -1;
}

int cuda_double_moore_penrose_inverse_host_memory( double *A, double **A_inv, int m, int n, int lda, double cutoff )
{
   hipError_t cudaStat = hipSuccess;

   int return_value = 0;
   const long unsigned int A_size =   get_size_(lda,       n , size_d);
   const long unsigned int A_i_size = get_size_(  m,       n , size_d);
   double *d_A;      // Initial matrix A
   double *d_A_inv;      // Initial matrix A

   cudaStat = hipMalloc ((void**)&d_A, A_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }
   cudaStat = hipMalloc ((void**)&d_A_inv, A_i_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }

   return_value = cuda_double_moore_penrose_inverse_device_memory( d_A, d_A_inv, m, n, lda, cutoff );

   A_inv[0] = (double*) malloc(A_size);
   cudaStat = hipMemcpy(A_inv[0], d_A_inv, A_size, hipMemcpyDeviceToHost);

   cleanup:
      if (d_A)     hipFree(d_A);
      if (d_A_inv) hipFree(d_A_inv);
      hipDeviceReset();
      return return_value;
}

int cuda_double_moore_penrose_inverse_device_memory( double *d_A, double *d_A_inv, int m, int n, int lda, double cutoff )
{
   hipError_t cudaStat = hipSuccess;

   // Start with checking the expected size of the problem.
   int memory_information = check_needed_memory (lda, m, n, sizeof(double));
   if ( memory_information == -1 )
   {
      std::cerr << "ERROR:\tNot enough GPU memory...\n";
      return ERROR_CUDA_MALLOC;
   }

   // Some values for the following calculations
   int return_value = 0;
   const int S_elements = min(m,n)*n;

   const long unsigned int S_size   = get_size_(  1, min(m,n), size_d);
   const long unsigned int S_i_size = get_size_(  n, min(m,n), size_d);
   const long unsigned int U_size   = get_size_(  m, min(m,n), size_d);
   const long unsigned int V_size   = get_size_(  n, min(m,n), size_d);
   const long unsigned int A_size   = get_size_(lda,       n , size_d);

   // Device pointers
//   double *d_A;      // Initial matrix A
   double *d_S;      // Vector of singular values
   double *d_S_inv;  // diag[ 1 / sigma_i ]
   double *d_U;      // SVD matrix U
   double *d_V;      // SVD matrix V
   double *d_VS;     // V * diag[1/sigma_i]
   double *U;
   d_S = d_S_inv = d_U = d_V = d_VS = U = NULL;

   // Allocate svd memory on GPU
   cudaStat = hipMalloc ((void**)&d_U, U_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }
//   cudaStat = hipMalloc ((void**)&d_A, A_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }
   cudaStat = hipMalloc ((void**)&d_V, V_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }
   cudaStat = hipMalloc ((void**)&d_S, S_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }

   if ( !memory_information )
   {
      cudaStat = hipMalloc ((void**)&d_S_inv, S_i_size);
      if (hipSuccess != cudaStat)
      {
         return_value = ERROR_CUDA_MALLOC;
         goto cleanup;
      }
   }

   // Copy initial matrix on GPU
//   cudaStat = hipMemcpy(d_A, A, A_size, hipMemcpyHostToDevice); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_COPY; goto cleanup; }

   // Compute the svd
   return_value = cuda_double_svd_device_memory( d_A, d_S, d_U, d_V, m, n, lda );
   if ( return_value )
   {
      std::cout << "svd_error = " << return_value << std::endl;
      goto cleanup;
   }

   if ( memory_information )
   {
      U = (double*) malloc (U_size);
      cudaStat = hipMemcpy(U, d_U, U_size, hipMemcpyDeviceToHost); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_COPY; goto cleanup; }
//      if (d_A) { hipFree(d_A); d_A = NULL; }
      if (d_U) { hipFree(d_U); d_U = NULL; }
      cudaStat = hipMalloc ((void**)&d_S_inv, S_i_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }
   }

   // Invert the singular values and write them on the diagonal of S_inv
   if ( n < THREADS_PER_BLOCK )
      cuda_invert_singular_values_device_memory<<<m,n>>> ( d_S, d_S_inv, cutoff, n, S_elements );
   else
   {
      int Number_of_blocks = (S_elements/THREADS_PER_BLOCK+1);  // +1 -> else last sigma might be skiped!
      cuda_invert_singular_values_device_memory<<<Number_of_blocks,THREADS_PER_BLOCK>>> ( d_S, d_S_inv, cutoff, n, S_elements );
   }

   // Free the vector S
   if (d_S) { hipFree(d_S); d_S = NULL; }

   // Allocate the memory for the product V*S_inv
   cudaStat = hipMalloc ((void**)&d_VS, V_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }

   // Compute the product
   // If "small svd is used we have to choose 't' for matrix d_V!
   cuda_dgemm('t', 'n', n, n, n, 1.0, d_V,  n, d_S_inv,   n, 0.0,    d_VS, n );

   // Free V since it is not needed anymore
   if (d_V) { hipFree(d_V); d_V = NULL; }
   if (d_S_inv) { hipFree(d_S_inv); d_S_inv = NULL; }

   // If U was shifted to main RAM write the respective values back
   // in initial dRAM and remove S_inv
   if ( memory_information)
   {
      cudaStat = hipMalloc ((void**)&d_U, U_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }
//      cudaStat = hipMalloc ((void**)&d_A, U_size); if (hipSuccess != cudaStat) return ERROR_CUDA_MALLOC;
      cudaStat = hipMemcpy(d_U, U, U_size, hipMemcpyHostToDevice); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_COPY; goto cleanup; }
      free(U); U = NULL;
   }

   // Allocate the memory for the final matrix A
   //cudaStat = hipMalloc ((void**)&d_A_inv, A_size); if (hipSuccess != cudaStat) { return_value = ERROR_CUDA_MALLOC; goto cleanup; }

   // And perform the final product
   cuda_dgemm('n', 't', n, m, n, 1.0, d_VS,   n,     d_U, lda, 0.0, d_A_inv, n );

   // Copy A_inverse back on host memory
//   A_inv[0] = (double*) malloc( A_size);
//   cudaStat = hipMemcpy(A_inv[0], d_A, A_size, hipMemcpyDeviceToHost);

// Free device memory and make sure that everyhting is freed!
   cleanup:
//      if (d_A)     hipFree(d_A);
      if (d_S)     hipFree(d_S);
      if (d_S_inv) hipFree(d_S_inv);
      if (d_U)     hipFree(d_U);
      if (d_V)     hipFree(d_V);
      if (d_VS)    hipFree(d_VS);
      if (U)       free(U);

//   hipDeviceReset();
   return return_value;
}
