
#include "../../cuda/include/cuda_matmult_double.hpp"

#include <hip/hip_runtime.h>
#include <hipblas.h>

void cuda_dgemm(
             char transa,
             char transb,
             const int &m,
             const int &n,
             const int &k,
             const double alpha,
             const double *A,
             const int &lda,
             const double *B,
             const int &ldb,
             const double beta,
             double *C,
             const int &ldc)
{
   hipblasDgemm( transa,
                transb,
                m,
                n,
                k,
                alpha,
                A,
                lda,
                B,
                ldb,
                beta,
                C,
                ldc);
}
